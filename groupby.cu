#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
enum reduction_op { max, min, sum, count, mean };

template<typename T>
struct Groupby
{
    T* key_columns[];
	int* key_starts;
	int* key_counts;
	int num_key_columns;
	int num_key_rows;

	T* value_columns[];
	int* output_index[];
	int num_value_columns;
	int num_value_rows;

	reduction_op ops[];
	int num_ops;

	T* output_keys;
	T* output_values;
	int num_output_rows;


};

//Launch reduction kernels for each column based on their specified operation
template <typename T>
void perform_operators(Groupby &input)
{
	//TODO: hashing here
	thrust::device_vector<T> d_hash_keys(input.num_key_rows);

	//create index array for sorting. 
	thrust::device_vector<int> d_i(input.num_key_rows);
	thrust::sequence(thrust::host, d_i.begin(), d_i.end()); 

	//sort by key, also sort values. The result can be used to sort the actual data arrays later
	thrust::sort_by_key(d_hash_keys.begin(), d_hash_keys.end(), d_i);

	//Find the position of first key for each group
	thrust::device_vector<int> d_key_starts(input.num_key_rows);
	thrust::sequence(d_key_starts.begin(), d_key_starts.end()); //this sequence represents the index for each key
	int new_end  = (thrust::unique_by_key(d_hash_keys.begin(), d_hash_keys.end(), d_key_starts.begin())).first - d_hash_keys.begin();
	//after unique_by_key, d_key_starts holds the start of each group.

	//setup output arrays
	input.num_output_rows = new_end;
	input.output_keys = new T[num_output_rows];
	input.output_values = new T[num_output_rows*input.num_value_columns];

	for (int i = 0; i<input.num_ops, i++){//i represents column of output
		//get this column of data. copy does [first, last) 
		int start = input.value_columns.begin() + i*input.num_value_rows;
		int end = input.value_columns.begin() + (i+1)*input.num_value_rows;
		thrust::device_vector<T> col(input.num_value_rows), sorted_col(input.num_value_rows);
		thrust::copy(input.value_columns.begin() + start, input.value_columns.begin() + end,col.begin());

		//the column is not sorted yet so use d_i to sort!
		thrust::copy_n(thrust::make_permutation_iterator(col.begin(), d_i.begin()), input.num_value_rows, sorted_col.begin());
		for (int j = 0; j < new_end; j++){ //iterate over the groups of keys... j = output row
			int start = d_key_starts[j];
			int end;
			if (j < new_end-1){
				end = d_key_starts[j+1];
			}else{
				end = col.end();
			}
			T val;
			switch(input.ops[i]){
				case max:
					val = *(thrust::max_element(col.begin() + start, col.begin() + end));
					break;
				case min:
					val = *((thrust::min_element(col.begin() + start, col.begin() + end));
					break;
				case sum:
					val = ((T)thrust::reduce(col.begin() + start, col.begin() + end));
					break;
				case count:
					val = (T) (end-start)+1;
					break;
				case mean:
					T count = (T) (end-start)+1;
					val = ((T)thrust::reduce(col.begin() + start, col.begin() + end))/((T)count);
					break;
			}
			output_values[i*num_output_rows+j] = val;
		}
	}
}