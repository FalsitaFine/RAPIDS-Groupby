#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

#include "cpuGroupby.h"
#include "groupby_hash.cuh"

// is there dynamic size constant memory?
__constant__ reductionType ops_c[512];


#include "groupby_hash_templates.cu"

size_t size_alignment(size_t size, size_t alignment)
{
  return (size + alignment - 1) / alignment;
}

void groupby_hash_GPU(const int* key_columns_h, int num_key_columns, int num_key_rows,
		      const int* value_columns_h, int num_value_columns, int num_value_rows,
		      reductionType* ops, int num_ops, int* output_keys, int* output_values, int &num_output_rows)
{
#ifdef DEBUG
  constexpr unsigned int BLOCKDIM = 512;
#else
  constexpr unsigned int BLOCKDIM = 1024;
#endif
  constexpr unsigned int HASH_TABLE_SIZE = 10003;
#ifndef TESLA
  constexpr unsigned int GRIDDIM = 40; 
#else
  constexpr unsigned int GRIDDIM = 112; 
#endif
  
  using Tval = int; // replace int with actual variable type if needed;
  
  // variableAllocating
  int* key_columns_d = NULL;
  int* value_columns_d = NULL;
  int* hash_key_idx_d = NULL;
  int* hash_count_d = NULL;
  int* hash_results_d = NULL;

  gpuErrchk(hipMalloc(&key_columns_d, sizeof(int)*num_key_columns*num_key_rows));
  gpuErrchk(hipMalloc(&value_columns_d, sizeof(int)*num_value_columns*num_value_rows));
  gpuErrchk(hipMalloc(&hash_key_idx_d, sizeof(int)*HASH_TABLE_SIZE));
  gpuErrchk(hipMalloc(&hash_count_d, sizeof(int)*HASH_TABLE_SIZE));
  gpuErrchk(hipMalloc(&hash_results_d, sizeof(Tval)*HASH_TABLE_SIZE*num_ops));
  
  // initialize values
  gpuErrchk(hipMemcpy(key_columns_d, key_columns_h, sizeof(int)*num_key_columns*num_key_rows, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(value_columns_d, value_columns_h, sizeof(int)*num_value_columns*num_value_rows, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ops_c), ops, sizeof(reductionType) * num_ops));
  initializeVariable<int><<<GRIDDIM, BLOCKDIM>>>(hash_key_idx_d, hash_count_d, hash_results_d, HASH_TABLE_SIZE, num_ops);
  gpuErrchk(hipDeviceSynchronize());

  // fill hash table
#ifndef PRIVATIZATION
  fillTable<int, int><<<GRIDDIM, BLOCKDIM>>>(key_columns_d, num_key_rows, num_key_columns,
					     value_columns_d, num_value_rows, num_value_columns,
					     hash_key_idx_d, hash_count_d, hash_results_d,
					     HASH_TABLE_SIZE, num_ops);
#else
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  size_t sharedMemPerBlock = deviceProp.sharedMemPerBlock;
  printf("Total amount of sharedmemory per block %u\n", sharedMemPerBlock);
# ifdef TESLA
  sharedMemPerBlock = 32 * 1024;
# endif
  size_t max_capacity = sharedMemPerBlock - sizeof(unsigned int);
  size_t s_len_table = max_capacity / (2*sizeof(int) + sizeof(Tval)*num_ops);
  size_t sharedMemorySize = 0;
  while (true) { // calculate the suitable length of shared memory table
    sharedMemorySize = size_alignment(2*sizeof(int)*s_len_table, sizeof(Tval)) * sizeof(int);
    sharedMemorySize += sizeof(Tval)*num_ops*s_len_table;
    if (sharedMemorySize < max_capacity)
      if (s_len_table % 2 == 1) break; // always make length an odd number to avoid serious collision
    --s_len_table;
  }
  printf("Length of Shared Table: %u\n", s_len_table);
  printf("Total extern shared memory: %u\n", sharedMemorySize);
  fillTable_privatization
    <int, int><<<GRIDDIM, BLOCKDIM, sharedMemorySize>>>(key_columns_d, num_key_rows,
							num_key_columns, value_columns_d,
							num_value_rows, num_value_columns,
							hash_key_idx_d, hash_count_d,
							hash_results_d, HASH_TABLE_SIZE,
							s_len_table, num_ops);
#endif
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  //shrink the hash table to output array
  //Create array of idices for hash table
  int *seq, *hashTable_idxs;
  hipMalloc((void**)&seq, HASH_TABLE_SIZE*sizeof(int)); //for hash index sequence  
  hipMalloc((void**)&hashTable_idxs, HASH_TABLE_SIZE*sizeof(int));  //for key indexs without -1   
  thrust::device_ptr<int> hash_d_seq = thrust::device_pointer_cast(seq); //for hash index sequence 
  thrust::device_ptr<int> hashTable_idxs_d = thrust::device_pointer_cast(hashTable_idxs); //for key indexs without -1 
  thrust::sequence(thrust::device, hash_d_seq, hash_d_seq + HASH_TABLE_SIZE); //fill hash index seq


  //copy hash idex of keys, removeing -1's which signify not used
//   copy_if(policy, index seq start, index seq end, hash keys for comparison, result containing idx to non -1's, comparator)
  auto newEnd = thrust::copy_if(thrust::device, hash_d_seq, hash_d_seq + HASH_TABLE_SIZE, hash_key_idx_d, hashTable_idxs_d, is_pos());
  
  num_output_rows = newEnd - hashTable_idxs_d;
  printf("%d output rows!\n", num_output_rows);

  int* output_key_columns_d = NULL;
  hipMalloc(&output_key_columns_d, sizeof(int)*num_key_columns*num_output_rows);
  copyUnique<int><<<GRIDDIM,BLOCKDIM>>>(hashTable_idxs, hash_key_idx_d,key_columns_d, output_key_columns_d, num_output_rows, num_key_columns, num_key_rows);

  int* output_value_columns_d = NULL;
  hipMalloc(&output_value_columns_d, sizeof(int)*num_value_columns*num_output_rows);
  copyValues<int><<<GRIDDIM,BLOCKDIM>>>(hashTable_idxs, hash_results_d,hash_count_d, value_columns_d, output_value_columns_d, num_output_rows, num_value_columns, num_value_rows, num_ops, HASH_TABLE_SIZE);

  gpuErrchk(hipDeviceSynchronize());

  // copy back

  gpuErrchk(hipMemcpy(output_keys,output_key_columns_d,sizeof(int)*num_key_columns*num_output_rows,hipMemcpyDeviceToHost)); 
  gpuErrchk(hipMemcpy(output_values,output_value_columns_d,sizeof(int)*num_value_columns*num_output_rows,hipMemcpyDeviceToHost)); 


  // free elements

  hipFree(key_columns_d);
  hipFree(value_columns_d);
  hipFree(hash_key_idx_d);
  hipFree(hash_count_d);
  hipFree(hash_results_d);
  hipFree(output_key_columns_d);
  hipFree(output_value_columns_d);
  hipFree(seq);
  hipFree(hashTable_idxs);
  
}
