#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cpuGroupby.h"
#include "groupby_hash_templates.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
} 

// is there dynamic size constant memory?
__constant__ reductionType ops_d[512];

void groupby_hash_GPU(const int* key_columns_h, int num_key_columns, int num_key_rows,
		      const int* value_columns_h, int num_value_columns, int num_value_rows,
		      reductionType* ops, int num_ops, int* output_keys, int* output_values, int &num_output_rows)
{
  constexpr unsigned int BLOCKDIM = 1024;
  constexpr unsigned int HASH_TABLE_SIZE = 1003;
  // variableAllocating
  int* key_columns_d = NULL;
  int* value_columns_d = NULL;
  int* hash_key_idx_d = NULL;
  int* hash_count_d = NULL;
  int* hash_results_d = NULL;
  gpuErrchk(hipMalloc(&key_columns_d, sizeof(int)*num_key_columns*num_key_rows));
  gpuErrchk(hipMalloc(&value_columns_d, sizeof(int)*num_value_columns*num_value_rows));
  gpuErrchk(hipMalloc(&hash_key_idx_d, sizeof(int)*HASH_TABLE_SIZE));
  gpuErrchk(hipMalloc(&hash_count_d, sizeof(int)*HASH_TABLE_SIZE));
  gpuErrchk(hipMalloc(&hash_results_d, sizeof(int)*HASH_TABLE_SIZE*num_ops));
  
  // initialize values
  gpuErrchk(hipMemcpy(key_columns_d, key_columns_h, sizeof(int)*num_key_columns*num_key_rows, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(value_columns_d, value_columns_h, sizeof(int)*num_value_columns*num_value_rows, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(ops_d), ops, sizeof(reductionType) * num_ops));
  initializeVariable<int><<<50, BLOCKDIM>>>(hash_key_idx_d, hash_count_d, hash_results_d, HASH_TABLE_SIZE, ops_d, num_ops);
  gpuErrchk(hipDeviceSynchronize());

  // fill hash table
  fillTable<int, int><<<50, BLOCKDIM>>>(key_columns_d, num_key_rows, num_key_columns,
					value_columns_d, num_value_rows, num_value_columns,
					hash_key_idx_d, hash_count_d, hash_results_d,
					HASH_TABLE_SIZE, ops_d, num_ops);
  gpuErrchk(hipDeviceSynchronize());

  // shrink the hash table to output array

  // copy back

  
}
