#include "hip/hip_runtime.h"
//
//  main.cpp
//  RAPIDS
//
//  Created by Aaron on 11/19/18.
//  Copyright © 2018 Aaron Nightingale. All rights reserved.
//
//  This is a simple CPU groupby class (currently only MAX is implemented)
//  one key column and one value column.

#include <iostream>
#include <time.h>
#include "cpuGroupby.h"

#include "groupby.cu"

using namespace std;
int main(int argc, const char * argv[]) {
        clock_t start, end;

        int num_rows = 100000;
        int num_key_cols = 2;
        int num_val_cols = 3;
        int num_distinct_keys = 3;
        if (argc == 2){
                num_rows = atoi(argv[1]);
        }else if(argc ==4){
                num_rows = atoi(argv[1]);
                num_key_cols = atoi(argv[2]);
                num_val_cols = atoi(argv[3]);
        }
        // Setting up the CPU groupby
        cpuGroupby slowGroupby(num_key_cols, num_val_cols, num_rows);

        slowGroupby.fillRand(num_distinct_keys, num_rows);

        start = clock();

        slowGroupby.groupby();

        end = clock(); 
        float cpu_duration = ((float)end-(float)start)/CLOCKS_PER_SEC; 

        // Insert GPU function calls here...
        int *gpu_output_keys, *gpu_output_values;
        int gpu_output_rows = 0;
        gpu_output_keys = (int *)malloc(slowGroupby.num_key_rows*slowGroupby.num_key_columns * sizeof(int));
        gpu_output_values = (int *)malloc(slowGroupby.num_value_rows*slowGroupby.num_value_columns * sizeof(int));

        start = clock();

        groupby_GPU(slowGroupby.key_columns, slowGroupby.num_key_columns,
                slowGroupby.num_key_rows, slowGroupby.value_columns, 
                slowGroupby.num_value_columns, slowGroupby.num_value_rows, 
                slowGroupby.ops, slowGroupby.num_ops,
                gpu_output_keys, gpu_output_values, gpu_output_rows); 
        slowGroupby.printGPUResults(gpu_output_keys, gpu_output_values);

        end = clock(); 
        float gpu_duration = ((float)end-(float)start)/CLOCKS_PER_SEC; 

        cout << "CPU time: " << cpu_duration << "s" << endl;
        cout << "GPU time: " << gpu_duration << "s" << endl;

        slowGroupby.validGPUResult(gpu_output_keys, gpu_output_values, gpu_output_rows);

        return 0;
}

/*
groupby( T* key_columns[], int num_key_columns, int num_key_rows,
        T* value_columns[], int num_value_columns, int num_value_rows,
        reduction_op ops[], int num_ops, T* output_keys[], T* output_values[]) {
    
    
}
*/
